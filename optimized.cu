
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>     
#include <sys/types.h>  
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>    
#include <errno.h>
#include <assert.h>

#define numThreads 1024


#define MAX_CHAR_PER_LINE 128

#define NONE 0
#define FIRST 1
#define LAST 2
#define BOTH 3 

double* file_read(char *filename,int  *numCoords,int  *numObjs) 
{
	double *objects;
	int     i, j, len;
    //ssize_t numBytesRead;
	int done=0; 
	FILE *infile;
	char *line, *ret;
	int   lineLen;

	//don't skip lines or attributes for this project
	int lines_to_skip=0; 
	int attr_to_skip=0;
	
	if ((infile = fopen(filename, "r")) == NULL) {
		fprintf(stderr, "Error: no such file (%s)\n", filename);
		return NULL;
	}

    /* first find the number of objects */
	lineLen = MAX_CHAR_PER_LINE;
	line = (char*) malloc(lineLen);
	assert(line != NULL);

	(*numCoords) = 0;

	while (fgets(line, lineLen, infile) != NULL) {
            /* check each line to find the max line length */
		while (strlen(line) == lineLen-1) {
                /* this line read is not complete */
			len = strlen(line);
			fseek(infile, -len, SEEK_CUR);

                /* increase lineLen */
			lineLen += MAX_CHAR_PER_LINE;
			line = (char*) realloc(line, lineLen);
			assert(line != NULL);

			ret = fgets(line, lineLen, infile);
			assert(ret != NULL);
		}

		if (strtok(line, " \t\n") != 0)
			(*numCoords)++;
	}

	(*numCoords)-=lines_to_skip;

	if((*numCoords)<=0)
	{
		fprintf(stderr, "Error: No objects found\n");
		return NULL;
	}

	rewind(infile);

	/*find the number of attributes*/  
	(*numObjs)=0;

	fgets(line, lineLen, infile);

	char * pch;
	pch=strtok(line, ",;");

	while (pch != NULL )
	{

		pch = strtok (NULL, ",;");
		(*numObjs)++;
	}

	if(attr_to_skip!=NONE)
	{
		(*numObjs)--;
		if(attr_to_skip==BOTH)
			(*numObjs)--;
	}

	rewind(infile);


    /* allocate space for objects and read all objects */
	len = (*numCoords) * (*numObjs);
	objects    = (double*)malloc( len * sizeof(double));
	assert(objects != NULL);



    /* read all objects */

	for(i=0;i<lines_to_skip;i++)
		fgets(line, lineLen, infile);

	i=0;
	j=0;

	while (fgets(line, lineLen, infile) != NULL) 
	{
		pch=strtok(line, ",;");
		while (pch != NULL && j<(*numObjs))
		{
			if(attr_to_skip%2==1 && j==0 && done==0)
			{
				done=1;
				pch = strtok (NULL, ",;");
				continue;                      
			}
			objects[i*(*numObjs)+j]=atof(pch);
			pch = strtok (NULL, ",;");
			j++;
		}
		i++;
		j=0;
		done=0;
	}

	assert(i == *numCoords);

	fclose(infile);
	free(line);


	return objects;
}

__global__ void median(double *M_d,double *w_d,int rows, int cols)
{
	__shared__ double w_ds[numThreads];
	int row = blockIdx.x;

	int loop = (cols - 1)/numThreads + 1;
	double sum = 0.0;
	for (int k=0;k<loop;k++){ //Μερικά Αθροίσματα
		if ( row < rows && (numThreads*k + threadIdx.x) < cols ){
			sum += M_d[row*cols + numThreads*k + threadIdx.x];
		}
	}
	w_ds[threadIdx.x] = sum;
	__syncthreads();

	for (unsigned int k=blockDim.x/2; k>0; k>>=1) { //Reduction
		if (threadIdx.x < k) {
			w_ds[threadIdx.x] += w_ds[threadIdx.x + k];
		}
		__syncthreads();
	}
	w_d[row] = w_ds[0]/cols;

} 

__global__ void MatrixVecFirst(double* M_d, double* x_d, double* z_d, double* w_d, int rows, int cols){
	__shared__ double Pvalue[numThreads];
	__shared__ double w_ds;

	int row = blockIdx.x;
	if (!threadIdx.x) w_ds = w_d[row];
	__syncthreads();

	int loop = (cols - 1)/numThreads + 1;
	double sum = 0.0;
	for (int k=0;k<loop;k++){ //Μερικά Αθροίσματα
		if ( row < rows && (numThreads*k + threadIdx.x) < cols ) 
		{
			sum+= (M_d[row*cols + numThreads*k + threadIdx.x] - w_ds)*x_d[numThreads*k + threadIdx.x];

		}         
	}
	Pvalue[threadIdx.x]=sum;
	__syncthreads();

for (unsigned int k=blockDim.x/2; k>0; k>>=1) { //Reduction
		if (threadIdx.x < k) {
			Pvalue[threadIdx.x] += Pvalue[threadIdx.x + k];
		}
		__syncthreads();
	}
	z_d[row] = Pvalue[0]; //Save result
} 



__global__ void MatrixVecSecond(double *M_d, double* x_d, double* z_d, double* k_d, double* w_d, int rows, int cols){
	int bx = blockIdx.x;
	int tx = threadIdx.x;

	int Row = bx*numThreads + tx;
	double sum = 0.0;
	if (Row < cols){ //Κάθε thread μια γραμμή
		for (int k=0;k<rows;k++){ //rows αντί για cols λόγω ανάστροφου
			sum += (M_d[k*cols + Row] - w_d[k])*z_d[k]; //διαφορετικός μο
		}
		k_d[Row] = sum;
	}	
}

__global__ void CalculateNorm(double* k_d, float* norm, int cols){
	__shared__ double k_ds[numThreads];

	int tx = threadIdx.x;
	int index = blockIdx.x*numThreads + tx;

	if (index < cols) k_ds[tx] = k_d[index]*k_d[index];
	else k_ds[tx] = 0;
	__syncthreads();

	for (unsigned int k=blockDim.x/2; k>0; k>>=1) {
		if (tx < k) {
			k_ds[tx] += k_ds[tx + k];
		}
		__syncthreads();
	}

	if (tx == 0) atomicAdd(norm,k_ds[tx]);
}

__global__ void DivideByNorm(double* k_d, float* norm, int cols){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cols) k_d[index] = k_d[index]/sqrt(*norm);
}

__global__ void CalculateEps(double* k_d, double* x_d, double* e_d, int cols){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cols) e_d[index] = k_d[index] - x_d[index];
}


int main(int argc, char **argv){
	int rows,cols;

	double* M_h = file_read(argv[1],&rows,&cols);
	double* x_h = (double*)malloc(cols*sizeof(double));
	double* z_h = (double*)malloc(rows*sizeof(double));
	double* k_h = (double*)malloc(cols*sizeof(double));
	float* e = (float*)malloc(sizeof(float));
	float* norm_h = (float*)malloc(sizeof(float));

	float* norm_d;
	float* eps;
	double* e_d;
	double* M_d;
	double* x_d;
	double* k_d;
	double* z_d;
	double* w_d;


	//Arxikopoiisi tou x_0
	for (int i=0;i<cols;i++) x_h[i] = 1;

	hipSetDevice(0);
	hipMalloc((void**) &M_d, rows*cols*sizeof(double));
	hipMalloc((void**) &x_d, cols*sizeof(double));
	hipMalloc((void**) &z_d, rows*sizeof(double));
	hipMalloc((void**) &k_d, cols*sizeof(double));
	hipMalloc((void**) &e_d, cols*sizeof(double));
	hipMalloc((void**) &w_d, rows*sizeof(double));
	hipMalloc((void**) &norm_d, sizeof(float));
	hipMalloc((void**) &eps, sizeof(float));

	hipMemcpy(M_d, M_h, rows*cols*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_h, cols*sizeof(double), hipMemcpyHostToDevice);

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	median<<<rows,numThreads>>>(M_d,w_d,rows,cols);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	for (int i=0;i<100;i++){ //Endeiktiki timi to 100 gia periptoseis mh sigklisis
		*norm_h = 0;
		*e = 0;
		hipMemcpy(norm_d, norm_h, sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(eps, e, sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(start);
		MatrixVecFirst<<<rows,numThreads>>>(M_d,x_d,z_d,w_d,rows,cols); //z_d = (M_d - w_d*e')*x_d

		MatrixVecSecond<<<(cols + numThreads -1)/numThreads,numThreads>>>(M_d,x_d,z_d,k_d, w_d, rows,cols); //k_d = (M_d - w_d*e')'*z_d

		CalculateNorm<<<(cols + numThreads-1)/numThreads,numThreads>>>(k_d, norm_d, cols); //norm(k_d)

		DivideByNorm<<<(cols + numThreads -1)/numThreads,numThreads>>>(k_d, norm_d, cols); //x_k+1 = k_d/norm(k_d)

		CalculateEps<<<(cols + numThreads -1)/numThreads,numThreads>>>(k_d, x_d, e_d, cols); //x_k+1 - x_k

		CalculateNorm<<<(cols + numThreads -1)/numThreads,numThreads>>>(e_d, eps, cols); // Ypologismos eps = ||x_k+1 - x_k||
		hipEventRecord(stop);
		hipEventSynchronize(stop);

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		elapsedTime += milliseconds;


		//Elegxos sigklisis
		hipMemcpy(e, eps, sizeof(float), hipMemcpyDeviceToHost);
		if (sqrt(*e) < 0.000001){
			hipMemcpy(k_h,k_d,cols*sizeof(double),hipMemcpyDeviceToHost);
			break;
		}

		//An den sigklinei, arxikopoiisi tou algorithmou me x_k+1
		hipMemcpy(x_d,k_d,cols*sizeof(double),hipMemcpyDeviceToDevice);
	}
	

	printf("Elapsed time: %f ms\n", elapsedTime);

	//Print to file
	FILE *f = fopen("results_opt.csv", "w");
	for (int i=0;i<cols;i++) fprintf(f,"%.7f%s", k_h[i],(i<cols-1)?",":"");
	fprintf(f,"\n");
	fclose(f);

	hipFree(M_d);
	hipFree(x_d);
	hipFree(z_d);
	hipFree(k_d);
	hipFree(w_d);
	hipFree(norm_d);
	hipFree(e_d);
	hipFree(eps);

	free(M_h);
	free(x_h);
	free(z_h);
	free(k_h);
	free(norm_h);
	free(e);
	return 0;
}
