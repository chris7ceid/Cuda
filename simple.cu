
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>     
#include <sys/types.h>  
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>    
#include <errno.h>
#include <assert.h>

#define numThreads 1024

#define MAX_CHAR_PER_LINE 128

#define NONE 0
#define FIRST 1
#define LAST 2
#define BOTH 3 

double* file_read(char *filename,int  *numCoords,int  *numObjs) 
{
  double *objects;
  int     i, j, len;
    //ssize_t numBytesRead;
  int done=0; 
  FILE *infile;
  char *line, *ret;
  int   lineLen;

	//don't skip lines or attributes for this project
  int lines_to_skip=0; 
  int attr_to_skip=0;

  if ((infile = fopen(filename, "r")) == NULL) {
    fprintf(stderr, "Error: no such file (%s)\n", filename);
    return NULL;
  }

    /* first find the number of objects */
  lineLen = MAX_CHAR_PER_LINE;
  line = (char*) malloc(lineLen);
  assert(line != NULL);

  (*numCoords) = 0;

  while (fgets(line, lineLen, infile) != NULL) {
            /* check each line to find the max line length */
    while (strlen(line) == lineLen-1) {
                /* this line read is not complete */
      len = strlen(line);
      fseek(infile, -len, SEEK_CUR);

                /* increase lineLen */
      lineLen += MAX_CHAR_PER_LINE;
      line = (char*) realloc(line, lineLen);
      assert(line != NULL);

      ret = fgets(line, lineLen, infile);
      assert(ret != NULL);
    }

    if (strtok(line, " \t\n") != 0)
      (*numCoords)++;
  }

  (*numCoords)-=lines_to_skip;

  if((*numCoords)<=0)
  {
    fprintf(stderr, "Error: No objects found\n");
    return NULL;
  }

  rewind(infile);

	/*find the number of attributes*/  
  (*numObjs)=0;

  fgets(line, lineLen, infile);

  char * pch;
  pch=strtok(line, ",;");

  while (pch != NULL )
  {

    pch = strtok (NULL, ",;");
    (*numObjs)++;
  }

  if(attr_to_skip!=NONE)
  {
    (*numObjs)--;
    if(attr_to_skip==BOTH)
      (*numObjs)--;
  }

  rewind(infile);


    /* allocate space for objects and read all objects */
  len = (*numCoords) * (*numObjs);
  objects    = (double*)malloc( len * sizeof(double));
  assert(objects != NULL);



    /* read all objects */

  for(i=0;i<lines_to_skip;i++)
   fgets(line, lineLen, infile);

 i=0;
 j=0;

 while (fgets(line, lineLen, infile) != NULL) 
 {
   pch=strtok(line, ",;");
   while (pch != NULL && j<(*numObjs))
   {
    if(attr_to_skip%2==1 && j==0 && done==0)
    {
      done=1;
      pch = strtok (NULL, ",;");
      continue;                      
    }
    objects[i*(*numObjs)+j]=atof(pch);
    pch = strtok (NULL, ",;");
    j++;
  }
  i++;
  j=0;
  done=0;
}

assert(i == *numCoords);

fclose(infile);
free(line);


return objects;
}

__global__ void medval(double *M_d,double *w_d,int rows, int cols)
{
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	double sum = 0.0;
	if (row < rows){
		for (int i=0;i<cols;i++){
			sum += M_d[row*cols + i];
		}
		w_d[row] = sum/cols;
	}	 
} 

__global__ void MatrixVecMul(double* M_d, double* x_d, double* z_d, double* w_d, int rows, int cols){
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	double sum = 0.0;
	if (row < rows){	
		for (int i=0;i<cols;i++){
			sum += (M_d[row*cols + i] - w_d[row])*x_d[i];
		}
		z_d[row] = sum;
	}
} 



__global__ void TMatrixVecMul(double *M_d, double* x_d, double* z_d, double* k_d, double* w_d, int rows, int cols, float* norm_d){
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	double sum = 0.0;
	if (row < cols){	
		for (int i=0;i<rows;i++){
			sum += (M_d[i*cols + row] - w_d[i])*z_d[i];
		}
		k_d[row] = sum;
		atomicAdd(norm_d,k_d[row]*k_d[row]);
	}

}

__global__ void DivideNorm(double* k_d, float* norm, int cols){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cols) k_d[index] = k_d[index]/sqrt(*norm);
}

__global__ void CalculateEps(double* k_d, double* x_d, double* e_d, float* eps, int cols){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index < cols){
   e_d[index] = k_d[index] - x_d[index];
   atomicAdd(eps,e_d[index]*e_d[index]);
 }
}


int main(int argc, char **argv){
	hipSetDevice(0);
	int rows,cols;
  double* M_h = file_read(argv[1],&rows,&cols);
  int size = rows*cols*sizeof(double);
  double* x_h = (double*)malloc(cols*sizeof(double));
  double* z_h = (double*)malloc(rows*sizeof(double));
  double* k_h = (double*)malloc(cols*sizeof(double));
  float* e = (float*)malloc(sizeof(float));
  float* norm_h = (float*)malloc(sizeof(float));

  float* norm_d;
  float* eps;
  double* e_d;
  double* M_d;
  double* x_d;
  double* k_d;
  double* z_d;
  double* w_d;



  for (int i=0;i<cols;i++) x_h[i] = 1;


  hipMalloc((void**) &M_d, size);
  hipMalloc((void**) &x_d, cols*sizeof(double));
  hipMalloc((void**) &z_d, rows*sizeof(double));
  hipMalloc((void**) &k_d, cols*sizeof(double));
  hipMalloc((void**) &e_d, cols*sizeof(double));
  hipMalloc((void**) &w_d, rows*sizeof(double));
  hipMalloc((void**) &norm_d, sizeof(float));
  hipMalloc((void**) &eps, sizeof(float));

  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);
  hipMemcpy(x_d, x_h, cols*sizeof(double), hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;
  hipEventRecord(start);
  medval<<<(rows - 1)/numThreads + 1,numThreads>>>(M_d,w_d,rows,cols);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  for (int i=0;i<100;i++){
    *norm_h = 0;
    *e = 0;
    hipMemcpy(norm_d, norm_h, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(eps, e, sizeof(float), hipMemcpyHostToDevice);
    if (i !=0){
     for (int k=0;k<cols;k++) x_h[k] = k_h[k];
       hipMemcpy(x_d, x_h, cols*sizeof(double), hipMemcpyHostToDevice);
   }
   hipEventRecord(start);
   MatrixVecMul<<<(rows - 1)/numThreads + 1,numThreads>>>(M_d,x_d,z_d,w_d,rows,cols);

   TMatrixVecMul<<<(cols - 1)/numThreads + 1,numThreads>>>(M_d,x_d,z_d,k_d, w_d, rows,cols,norm_d);

   DivideNorm<<<(cols - 1)/numThreads + 1,numThreads>>>(k_d, norm_d, cols);

   CalculateEps<<<(cols - 1)/numThreads + 1,numThreads>>>(k_d, x_d, e_d, eps, cols);
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);
   elapsedTime += milliseconds;

   hipMemcpy(k_h,k_d,cols*sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(e, eps, sizeof(float), hipMemcpyDeviceToHost);

   if (sqrt(*e) < 0.000001) break;
 }
 printf("Elapsed Time: %f ms\n", elapsedTime);
 FILE *f = fopen("results_simple.csv", "w");

 for (int i=0;i<cols;i++) fprintf(f,"%.7f%s", k_h[i],(i<cols-1)?",":"");
  fprintf(f,"\n");
fclose(f);

hipFree(M_d);
hipFree(x_d);
hipFree(z_d);
hipFree(k_d);
hipFree(w_d);
hipFree(norm_d);
hipFree(e_d);
hipFree(eps);

free(M_h);
free(x_h);
free(z_h);
free(k_h);
free(norm_h);
free(e);
return 0;
}

